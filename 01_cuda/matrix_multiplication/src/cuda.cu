#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "timer.h"

#include <stdio.h>
#include <iostream>
#include <exception>
#include <cmath>

using namespace std;

// Define a kernel function, which is the entry point
// for execution on the GPU
__global__ void matrix_multiplication(float *m1, float *m2, float *result, unsigned int m1_rows, unsigned int m1_cols, unsigned int m2_cols)
{
    // Get the row and column of the current element
    unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;
    unsigned int j = blockIdx.y*blockDim.y + threadIdx.y;

    // Return if the current element is out of bounds
    if (i >= m1_rows || j >= m2_cols) {
        return;
    }

    // Compute the dot product of the row of m1 and the column of m2
    float value = 0;
    for (unsigned int k = 0; k < m1_cols; k++) {
        value += m1[i * m1_cols + k] * m2[k * m2_cols + j];
    }

    // Store the result in the output matrix
    result[i * m2_cols + j] = value;
}

template <unsigned int BLOCK_SIZE>
__global__ void matrix_block_multiplication(float *m1, float *m2, float *result, unsigned int m1_rows, unsigned int m1_cols, unsigned int m2_cols)
{
    // Get the row and column of the current element
    unsigned int ti = threadIdx.x;
    unsigned int i = blockIdx.x*blockDim.x + ti;
    unsigned int tj = threadIdx.y;
    unsigned int j = blockIdx.y*blockDim.y + tj;

    __shared__ float m1_s[BLOCK_SIZE][BLOCK_SIZE];
    __shared__ float m2_s[BLOCK_SIZE][BLOCK_SIZE];

    float r = 0.f;
    for (unsigned int k=0; k < m1_cols/BLOCK_SIZE; k++) {
        m1_s[ti][tj] = m1[i*m1_cols + k*BLOCK_SIZE + tj];
        m2_s[ti][tj] = m2[(k*BLOCK_SIZE + ti)*m2_cols + j];
        __syncthreads();
        for (unsigned int l=0; l<BLOCK_SIZE; ++l) {
            r += m1_s[ti][l]*m2_s[l][tj];
        }
        __syncthreads();
    }
    result[i * m2_cols + j] = r;
}

template <unsigned int BLOCK_SIZE>
vector<float> cuda_block_multiplication_template(const std::vector<float>& m1,
                                    const std::vector<float>& m2,
                                    unsigned int m1_rows,
                                    unsigned int m1_cols,
                                    unsigned int m2_cols,
                                    bool use_shared_memory)
{
    string name = "CUDA Block Multiplication [" + std::to_string(BLOCK_SIZE) + "], use shared memory: ";
    if (use_shared_memory) {
        name += "true";
    } else {
        name += "false";
    }
    auto& timer = util::timers.gpu_add(name);
    // Allocate memory on the host
    vector<float> result(m1_rows * m2_cols);
    // Allocate memory on the device
    float *d_m1, *d_m2, *d_result;
    hipMalloc(&d_m1, m1_rows * m1_cols * sizeof(float));
    hipMalloc(&d_m2, m1_cols * m2_cols * sizeof(float));
    hipMalloc(&d_result, m1_rows * m2_cols * sizeof(float));
    // Copy data from host to device
    hipMemcpy(d_m1, m1.data(), m1_rows * m1_cols * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_m2, m2.data(), m1_cols * m2_cols * sizeof(float), hipMemcpyHostToDevice);
    // // sync cuda device
    // hipDeviceSynchronize();
    // Define grid and block size
    dim3 grid(ceil((float)m1_rows/BLOCK_SIZE), ceil((float)m2_cols/BLOCK_SIZE), 1);
    dim3 block(BLOCK_SIZE, BLOCK_SIZE, 1);
    if (use_shared_memory) {
        matrix_block_multiplication<BLOCK_SIZE><<<grid, block>>>(d_m1, d_m2, d_result, m1_rows, m1_cols, m2_cols);
    } else {
        matrix_multiplication<<<grid, block>>>(d_m1, d_m2, d_result, m1_rows, m1_cols, m2_cols);
    }
    // Launch kernel
    // // sync cuda device
    // hipDeviceSynchronize();
    // Copy data from device to host
    hipMemcpy(result.data(), d_result, m1_rows * m2_cols * sizeof(float), hipMemcpyDeviceToHost);
    // Free memory on the device
    hipFree(d_m1);
    hipFree(d_m2);
    hipFree(d_result);
    timer.stop();
    return result;
}


vector<float> cuda_multiplication(const std::vector<float>& m1,
                                    const std::vector<float>& m2,
                                    unsigned int m1_rows,
                                    unsigned int m1_cols,
                                    unsigned int m2_cols,
                                    unsigned int block_size,
                                    bool use_shared_memory)
{
    switch (block_size) {
        case 4:
            return cuda_block_multiplication_template<4>(m1, m2, m1_rows, m1_cols, m2_cols, use_shared_memory);
        case 8:
            return cuda_block_multiplication_template<8>(m1, m2, m1_rows, m1_cols, m2_cols, use_shared_memory);
        case 16:
             return cuda_block_multiplication_template<16>(m1, m2, m1_rows, m1_cols, m2_cols, use_shared_memory);
        case 32:
        default:
             return cuda_block_multiplication_template<32>(m1, m2, m1_rows, m1_cols, m2_cols, use_shared_memory);
    }
}